#include "hip/hip_runtime.h"

#include <stdio.h>

#include "GPUManager/GPUCustomTypes.h"
#include "GPUManager/CUDA-Complex.cuh"
#include "DblReggeHelper.cuh"
//#include "AMPTOOLS_AMPS/DblReggeHelper.cuh"

__global__ void
DblReggeMod_kernel(GPU_AMP_PROTO, GDouble S0, GDouble b_par, GDouble c0, GDouble c1, GDouble c2, GDouble n0, GDouble n1, GDouble n2, GDouble a0, GDouble a1, GDouble a2, GDouble a3, GDouble a4, int fastParticle, int charge){

	int iEvent = GPU_THIS_EVENT;

	// here we need to be careful to index the user-defined
	// data with the proper integer corresponding to the
	// enumeration in the C++ header file

	//user vars as defined in enum in header:


	GDouble s12 = GPU_UVARS(0);
	GDouble s23 = GPU_UVARS(1);
	GDouble t1 = GPU_UVARS(2);
//	GDouble t2 = GPU_UVARS(3);
	GDouble s = GPU_UVARS(4);
	GDouble u3 = GPU_UVARS(5);
	GDouble beamM2 = GPU_UVARS(6);
	GDouble p1M2 = GPU_UVARS(7);
	GDouble p2M2 = GPU_UVARS(8);
	GDouble recoilM2 = GPU_UVARS(9);
	GDouble phi = GPU_UVARS(12);
//	GDouble up2 = GPU_UVARS(11);



	WCUComplex amp =  GPU_calcAmplitude(phi,s, s12, s23, t1, u3, S0, b_par,c0,c1,c2,n0,n1,n2,a0,a1,a2,a3,a4,beamM2, p1M2, p2M2, recoilM2, fastParticle, charge);

//if((amp.Re() + amp.Im()) > 35)
//{
//printf( "amp: %f \n", amp);
//printf( "u3: %f \n", u3);
//printf( "t1: %f \n", t1);
//printf( "s23: %f \n", s23);
//printf( "s12: %f \n", s12);
//}	

	pcDevAmp[iEvent] = amp;
}



void GPUDblReggeMod_exec( dim3 dimGrid, dim3 dimBlock, GPU_AMP_PROTO,
		GDouble S0, GDouble b_par, GDouble c0, GDouble c1, GDouble c2, GDouble n0, GDouble n1, GDouble n2, GDouble a0, GDouble a1, GDouble a2, GDouble a3, GDouble a4,int fastParticle, int charge  )
{

	DblReggeMod_kernel<<< dimGrid, dimBlock >>>( GPU_AMP_ARGS, S0, b_par, c0,c1,c2,n0,n1,n2, a0,a1,a2,a3,a4,fastParticle, charge );
}

