#include "hip/hip_runtime.h"

#include <stdio.h>

#include "GPUManager/GPUCustomTypes.h"
#include "GPUManager/CUDA-Complex.cuh"
#include "DblReggeHelper.cuh"
//#include "AMPTOOLS_AMPS/DblReggeHelper.cuh"

__global__ void
DblReggeMod_kernel(GPU_AMP_PROTO, GDouble b_par, GDouble c0, GDouble c1, GDouble c2, GDouble n0, GDouble n1, GDouble n2, GDouble d10, GDouble d11, GDouble d12,GDouble d20, GDouble d21,GDouble d22, GDouble aPrime, GDouble a0, GDouble S0, int fastParticle, int charge){

	int iEvent = GPU_THIS_EVENT;

	// here we need to be careful to index the user-defined
	// data with the proper integer corresponding to the
	// enumeration in the C++ header file

	//user vars as defined in enum in header:


	GDouble s12 = GPU_UVARS(0);
	GDouble s23 = GPU_UVARS(1);
	GDouble t1 = GPU_UVARS(2);
//	GDouble t2 = GPU_UVARS(3);
	GDouble s = GPU_UVARS(4);
	GDouble u3 = GPU_UVARS(5);
	GDouble beamM2 = GPU_UVARS(6);
	GDouble p1M2 = GPU_UVARS(7);
	GDouble p2M2 = GPU_UVARS(8);
	GDouble recoilM2 = GPU_UVARS(9);
//	GDouble up1 = GPU_UVARS(10);
//	GDouble up2 = GPU_UVARS(11);



	WCUComplex amp =  GPU_calcAmplitude(s, s12, s23, t1, u3, b_par, c0,c1,c2,n0,n1,n2, d10, d11,d12,d20,d21,d22,aPrime,a0, S0,beamM2, p1M2, p2M2, recoilM2, fastParticle, charge);

//if((amp.Re() + amp.Im()) > 35)
//{
//printf( "amp: %f \n", amp);
//printf( "u3: %f \n", u3);
//printf( "t1: %f \n", t1);
//printf( "s23: %f \n", s23);
//printf( "s12: %f \n", s12);
//}	

	pcDevAmp[iEvent] = amp;
}



void GPUDblReggeMod_exec( dim3 dimGrid, dim3 dimBlock, GPU_AMP_PROTO,
		GDouble b_par,GDouble c0, GDouble c1, GDouble c2, GDouble n0, GDouble n1, GDouble n2, GDouble d10, GDouble d11,GDouble d12,GDouble d20, GDouble d21, GDouble d22, GDouble aPrime, GDouble a0, GDouble S0, int fastParticle, int charge  )
{

	DblReggeMod_kernel<<< dimGrid, dimBlock >>>( GPU_AMP_ARGS, b_par,c0,c1,c2,n0,n1,n2,d10,d11,d12,d20,d21,d22,aPrime,a0,S0,fastParticle, charge );
}

